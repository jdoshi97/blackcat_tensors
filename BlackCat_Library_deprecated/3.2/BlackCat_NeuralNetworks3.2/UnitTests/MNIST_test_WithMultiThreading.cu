
#include <hip/hip_runtime.h>
//#include "../BlackCat_NeuralNetworks.h"
//#include "BlackCat_Tensors.h"
//
//#include <fstream>
//#include <iostream>
//#include <string>
//#include <omp.h>
//#include <mutex>				//ADD THIS FOR MUTXES (multi-thread locking)
//#include <thread>
//using BC::vec;
//typedef std::vector<vec> data;
//
//namespace BC {
//namespace MNIST_Test_multi_thread {
//typedef vec tensor;
//
//tensor expandOutput(int val, int total) {
//	//Convert a value to a 1-hot output vector
//	tensor out(total);
//	out.zero();
//	out[val] = (fp_type) 1.0;
//	return out;
//}
//
//tensor& normalize(tensor& tens, fp_type max, fp_type min) {
//	//generic feature scaling (range of [0,1])
//	tens -= Scalar<fp_type, BC::ml>(min);
//	tens /= Scalar<fp_type, BC::ml>(max - min);
//
//	return tens;
//}
//
//void generateAndLoad(data& input_data, data& output_data, std::ifstream& read_data, int MAXVALS) {
//	unsigned vals = 0;
//
//	//load data from CSV (the first number if the correct output, the remaining 784 columns are the pixels)
//	while (read_data.good() && vals < MAXVALS) {
//		std::string output;
//		std::getline(read_data, output, ',');
//		int out = std::stoi(output);
//
//		tensor input(784);
//		input.read(read_data, false);
//		output_data.push_back(expandOutput(out, 10));
//
//		normalize(input, 255, 0);
//		input_data.push_back(input);
//		++vals;
//	}
//	std::cout << " return -- finished creating data set " << std::endl;
//}
////
//template<int size>
//struct linear_map {
//
//	unsigned int pthread_id[size];
//	std::mutex mtx;
//
//	void reset() {
//		mtx.lock();
//		curr_size = 0;
//		for (int i = 0; i < size; ++i) {
//			pthread_id[i] = 0.0;
//		}
//		mtx.unlock();
//	}
//
//	int curr_size = 0;
//	int getID(unsigned int ptID) {
//		for (int i = 0; i < curr_size; ++i) {
//			if (pthread_id[i] == ptID) {
//				return i;
//			}
//		}
//
//		return add(ptID);
//	}
//
//	int add(unsigned int ptID) {
//		if (curr_size == size) {
//			throw std::invalid_argument("threads given exceeds limit");
//		}
//
//		mtx.lock();
//		pthread_id[curr_size] = ptID;
//		curr_size++;
//		mtx.unlock();
//		return curr_size - 1;
//	}
//};
//
//
//void train(NeuralNetwork<FeedForward, FeedForward, OutputLayer>& network, const data& inputs, const data& outputs, int start_range, int end_range) {
//	for (int i = start_range; i < end_range; ++i) {
//		network.forwardPropagation(inputs[i]);
//		network.backPropagation(outputs[i]);
//	}
//}
//
//void thread_test(int n) {
//	std::cout << n << std::endl;
//}
//
//int percept_MNIST() {
//
//	const int TRAINING_EXAMPLES = 400;
//	const int TRAINING_ITERATIONS = 10;
//	const int NUMB_THREADS = 2;
//
//	const int BATCH_SIZE = 100 / NUMB_THREADS; //must be divisible
//
//
//    omp_set_num_threads(NUMB_THREADS);
//    linear_map<NUMB_THREADS> thread_map;
//
//
//	//Generate the layers (params are: inputs, outputs)
//	FeedForward f1(784, 250);
//	FeedForward f2(250, 10);
//	OutputLayer o3(10);
//
//	//Create the neural network
//	auto network = generateNetwork(f1, f2, o3);									//Create the current network
//	std::vector<decltype(network)> networkSet(NUMB_THREADS, network);		//Create n copies where n = numb threads used - 1 (as we already created an nn)
////	linear_map<NUMB_THREADS> thread_map;										//create a linear map -> this is just a small object to convert pthreads into integers
//
//	//standard data handling code (same as other examples)
//	data inputs;
//	data outputs;
//
//	data testInputs;
//	data testOutputs;
//
//	//load data
//	std::cout << "loading data..." << std::endl << std::endl;
//	std::ifstream in_stream("///home/joseph///Downloads///train.csv");
//	std::string tmp;
//	std::getline(in_stream, tmp, '\n'); //remove headers
//
//	//Load training examples (taken from kaggle digit recognizer train.csv)
//	std::cout << " generating and loading data from csv to tensors" << std::endl;
//	generateAndLoad(inputs, outputs, in_stream, TRAINING_EXAMPLES);
//	in_stream.close();
//
//	for (int i = 0; i < TRAINING_ITERATIONS; ++i) {
//		std::cout << " iteration = " << i << std::endl;
//		//make sure BATCH_SIZE is a divider of n_inputs
//		for (int n = 0; n < inputs.size(); n+= BATCH_SIZE) {
//
//#pragma omp parallel for schedule(static)
//
//		for (int j = n; j < BATCH_SIZE + n; ++j) {
//			int netId = thread_map.getID(pthread_self());
//			networkSet[netId].forwardPropagation(inputs[j]);
//			networkSet[netId].backPropagation(outputs[j]);
//		}
//
//#pragma omp barrier
//		for (int k = 0; k < NUMB_THREADS; ++k) {
//
//			network.updateWeights(networkSet[k]);
//			networkSet[k].clearBPStorage();
//		}
//		for (int k = 0; k < NUMB_THREADS; ++k) {
//			networkSet[k].fastCopy(network);
//		}
//		thread_map.reset();
//	}
//	}
//	std::cout << "\n \n \n " << std::endl;
//	std::cout << " testing... " << std::endl;
//
//	for (int i = 0; i < 10; ++i) {
//		std::cout << " output " << std::endl;
//		outputs[i].print();
//		std::cout << "prediction " << std::endl;
//		network.forwardPropagation(inputs[i]).print();
//		std::cout << "-----------------------------------------------------------------------------------------------------------" << std::endl;
//
//	}
//
//	std::cout << " done training " << std::endl;
//	return 0;
//}
//}
//}
//
////int main() {
////	BC::MNIST_Test_multi_thread::percept_MNIST();
////	std::cout << "success" << std::endl;
////}
////