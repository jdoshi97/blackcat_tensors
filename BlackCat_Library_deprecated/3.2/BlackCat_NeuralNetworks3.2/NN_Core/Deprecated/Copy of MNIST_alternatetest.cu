
#include <hip/hip_runtime.h>
//#include "../BlackCat_NeuralNetworks.h"
//#include "../NN_Core/NeuralNetworkTest.h"
//#include "BlackCat_Tensors.h"
//
//#include <fstream>
//#include <iostream>
//#include <string>
//#include <omp.h>
//using BC::vec;
//typedef std::vector<vec> data;
//
//namespace BC {
//namespace MNIST_altTest {
//typedef vec tensor;
//
//tensor expandOutput(int val, int total) {
//	//Convert a value to a 1-hot output vector
//	tensor out(total);
//	out.zero();
//	out[val] = (fp_type)1.0;
//	return out;
//}
//
//tensor&  normalize(tensor& tens, fp_type max, fp_type min) {
//	//generic feature scaling (range of [0,1])
//	tens -= Scalar<fp_type, BC::ml>(min);
//	tens /= Scalar<fp_type, BC::ml>(max - min);
//
//	return tens;
//}
//bool correct(const vec& hypothesis, const vec& output) {
//	int h_id = 0;
//	int o_id = 0;
//
//	double h_max = hypothesis.data()[0];
//	double o_max = output.data()[0];
//
//	for (int i = 1; i < hypothesis.size(); ++i) {
//		if (hypothesis.data()[i] > h_max) {
//			h_max = hypothesis.data()[i];
//			h_id = i;
//		}
//		if (output.data()[i] > o_max) {
//			o_max = output.data()[i];
//			o_id = i;
//		}
//	}
//	return h_id == o_id;
//}
//
//
//
//void generateAndLoad(data& input_data, data& output_data, std::ifstream& read_data, int MAXVALS) {
//	unsigned vals = 0;
//
//	//load data from CSV (the first number if the correct output, the remaining 784 columns are the pixels)
//	while (read_data.good() && vals < MAXVALS) {
//		std::string output;
//		std::getline(read_data, output, ',');
//		int out = std::stoi(output);
//
//		tensor input(784);
//		input.read(read_data, false);
//		output_data.push_back(expandOutput(out, 10));
//
//		normalize(input, 255, 0);
//		input_data.push_back(input);
//		++vals;
//	}
//	std::cout << " return -- finished creating data set " << std::endl;
//}
//
//int percept_MNIST() {
//
//	const int TRAINING_EXAMPLES = 2000;
//	const int TRAINING_ITERATIONS = 10;
//
//	//Generate the layers (params are: inputs, outputs)
//	FeedForward f1(784, 250);
//	FeedForward f2(250, 10);
//	OutputLayer o3(10);
//	//Create the neural network
//	auto network = generateNetworkTest(f1, f2, o3);
//
//
//
//	data inputs;
//	data outputs;
//
//	data testInputs;
//	data testOutputs;
//
//	//load data
//	std::cout << "loading data..." << std::endl << std::endl;
//	std::ifstream in_stream("///home/joseph///Downloads///train.csv");
//	std::string tmp; std::getline(in_stream, tmp, '\n'); //remove headers
//
//	//Load training examples (taken from kaggle digit recognizer train.csv)
//	std::cout << " generating and loading data from csv to tensors" << std::endl;
//	generateAndLoad(inputs, outputs, in_stream, TRAINING_EXAMPLES);
//	in_stream.close();
//
//
//	//Train
//	float t;
//	t = omp_get_wtime();
//		printf("\n Calculating... BC_NN training time \n");
//
//
//
//	std::cout << " training..." << std::endl;
//
//	for (int i = 0; i < TRAINING_ITERATIONS; ++i) {
//		std::cout << " iteration =  " << i << std::endl;
//		for (int j = 0; j < inputs.size(); ++j) {
//			network.forwardPropagation(inputs[j]);
//			network.backPropagation(outputs[j]);
//
//			//this is just the batch size
//			if (j % 100 == 0) {
////			std::cout << " update: " << j / 100 << std::endl;
//			network.updateWeights();
//			network.clearBPStorage();
//			}
//		}
//	}
//
//
//	t = omp_get_wtime() - t;
//	printf("It took me %f clicks (%f seconds).\n", t, ((float) t));
//	std::cout << "success " << std::endl;
//
//	float correct_ = 0;
//	for (int i = 0; i < inputs.size(); ++i) {
//		if (correct(network.forwardPropagation(inputs[i]), outputs[i])) {
//			++correct_;
//		}
//	}
//	std::cout << " correct: " << correct_/inputs.size()  <<std::endl;
//
//
//	std::cout << "\n \n \n " << std::endl;
//	std::cout << " testing... " << std::endl;
//
//	for (int i = 0; i < 10; ++i) {
//		std::cout << " output " << std::endl;
//		outputs[i].print();
//		std::cout << "prediction " << std::endl;
//		network.forwardPropagation(inputs[i]).print();
//		std::cout << "-----------------------------------------------------------------------------------------------------------" <<std::endl;
//
//	}
//
//
//	std::cout << " done training " << std::endl;
//	return 0;
//}
//}
//}
//
//
//int main() {
//	BC::MNIST_altTest::percept_MNIST();
//	std::cout << "success" << std::endl;
//}
//